
#include <hip/hip_runtime.h>
#define THREADS 256
#define WIDTH 16  // shared memory width
#define HEIGHT 16 // shared memory height

///////////////////////////////
///////////////////////////////

extern "C" __global__ void
backprop(float *input_cuda,
                       float *output_hidden_cuda,
                       float *input_hidden_cuda,
                       float *hidden_partial_sum,
                       int in,
                       int hid) {

    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int index = (hid + 1) * HEIGHT * by + (hid + 1) * ty + tx + 1 + (hid + 1);

    int index_in = HEIGHT * by + ty + 1;

    __shared__ float input_node[HEIGHT];
    __shared__ float weight_matrix[HEIGHT][WIDTH];

    // Unsafe access;
    if (tx == 0) {
        input_node[ty] = input_cuda[index_in];
    }

    __syncthreads();

    // Unsafe access;
    weight_matrix[ty][tx] = input_hidden_cuda[index];

    __syncthreads();

    weight_matrix[ty][tx] = weight_matrix[ty][tx] * input_node[ty];

    __syncthreads();

    for (int i = 1; i <= __log2f(HEIGHT); i++) {

        int power_two = __powf(2, i);

        if (ty % power_two == 0) {
            weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + power_two / 2][tx];
        }

        __syncthreads();
    }

    input_hidden_cuda[index] = weight_matrix[ty][tx];

    __syncthreads();

    if (tx == 0) {
        hidden_partial_sum[by * hid + ty] = weight_matrix[tx][ty];
    }
}


