
#include <hip/hip_runtime.h>


extern "C" __global__ void hotspot3d_checked(float *p, float *tIn, float *tOut, float sdc,
                            int nx, int ny, int nz,
                            float ce, float cw,
                            float cn, float cs,
                            float ct, float cb,
                            float cc) {
    float amb_temp = 80.0;

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int size = nx * ny * nz;
    int c = i + j * nx;
    int xy = nx * ny;

    int W = (i == 0) ? c : c - 1;
    int E = (i == nx - 1) ? c : c + 1;
    int N = (j == 0) ? c : c - nx;
    int S = (j == ny - 1) ? c : c + nx;

    float temp1, temp2, temp3;
    if (c +xy < size) {
        temp1 = temp2 = tIn[c];
        temp3 = tIn[c + xy];
        tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S] + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
    }
    c += xy;
    W += xy;
    E += xy;
    N += xy;
    S += xy;

    for (int k = 1; k < nz - 1; ++k) {
        if (c + xy >= size || N >= size || E >= size || W >= size || S >= size)
            return;
        temp1 = temp2;
        temp2 = temp3;
        temp3 = tIn[c + xy];
        tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S] + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
        c += xy;
        W += xy;
        E += xy;
        N += xy;
        S += xy;
    }
    temp1 = temp2;
    temp2 = temp3;
    if (c + xy >= size || N >= size || E >= size || W >= size || S >= size)
        return;
    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S] + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
}
