
#include <hip/hip_runtime.h>
#define NUM_THREADS_3 16

extern "C" __global__ void autocov_checked(float *x, int k, int size, float *res) {
    __shared__ float cache[NUM_THREADS_3][NUM_THREADS_3];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int t = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < size - t) {
        cache[threadIdx.y][threadIdx.x] = x[i] * x[i + t];
    }
    __syncthreads();

    // Perform tree reduction;
    i = NUM_THREADS_3 / 2;
    while (i > 0) {
        if (t < k && threadIdx.x < i) {
            cache[threadIdx.y][threadIdx.x] += cache[threadIdx.y][threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }
    if (threadIdx.x == 0 && t < k) {
        atomicAdd(&res[t], cache[threadIdx.y][0] / size);
    }
}
