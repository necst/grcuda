#include "hip/hip_runtime.h"
#include "b22.cuh"
#include <cstdio>
#include <time.h>

//#define NGPU 4
#define GPU0 0
#define GPU1 1

#define A(i,j,N) A[(i)*N+(j)]
#define U(i,j,N) U[(i)*N+(j)]
#define L(i,j,N) L[(i)*N+(j)]
// #define LT(i,j,N) L[(i)*N+(j)]
// #define A_hw(i,j,N) A_hw[(i)*N+(j)]
// #define U_hw(i,j,N) U_hw[(i)*N+(j)]
// #define L_hw(i,j,N) L_hw[(i)*N+(j)]

// Multi-kernel GPU implementation
// L matrix is transposed -> we obtain 2 upper triangular matrix
__global__ void updateU(const float *A, float *U, float *L, const int offset, const int max, const int dim, const int row){
	int j = offset + blockIdx.x * blockDim.x + threadIdx.x;
	//int id = row * dim + j;
	if(j<max && j<dim && j >= row){
		U(row,j,dim) = A(row,j,dim);
		for(int k = 0; k<row; k++)
			U(row,j,dim) -= L(k,row,dim)*U(k,j,dim);
	}
}

__global__ void updateL(const float *A, float *U, float *L, const int offset, const int max, const int dim, const int col){
	int i = offset + blockIdx.x * blockDim.x + threadIdx.x;
	//int id = row * dim + j;
	if(i<max && i<dim && i>=col){
		L(col,i,dim) = A(i,col,dim);
		for(int k = 0; k<col; k++)
			L(col,i,dim) -= L(k,i,dim)*U(k,col,dim);
		L(col,i,dim) /= U(col,col,dim);
	}
}

/*
// Multi-kernel SW implementation
// L matrix is transposed -> we obtain 2 upper triangular matrix
void SWupdateUT(float A[DIM][DIM], float U[DIM][DIM], float LT[DIM][DIM], int dim, int row){
	for(int j = row; j<dim; j++){
		U[row][j] = A[row][j];
		for(int k = 0; k<row; k++)
			U[row][j] -= LT[k][row]*U[k][j];
	}
}

void SWupdateLT(float A[DIM][DIM], float U[DIM][DIM], float LT[DIM][DIM], int dim, int row){
	for(int j = row; j<dim; j++){
		LT[row][j] = A[j][row];
		for(int k = 0; k<row; k++)
			LT[row][j] -= LT[k][j]*U[k][row];
		LT[row][j] /= U[row][row];
	}
}

// initialize A with random numbers in [-99, 99]
void initializeA(float A[DIM][DIM], int dim, int seed){
	srand(seed);
	for(int i = 0; i<dim; i++)
		for(int j = 0; j<dim; j++){
			A[i][j] = rand()%100 + 1;
			if(rand()%2)
				A[i][j] *= -1;
		}
}
*/
  
void Benchmark22::alloc(){
	hipMallocManaged(&A, long(N)*long(N)*sizeof(float));
	hipMallocManaged(&U, long(N)*long(N)*sizeof(float));
	hipMallocManaged(&L, long(N)*long(N)*sizeof(float));

	s = (hipStream_t *)malloc(sizeof(hipStream_t) * NGPU);
	for (int i = 0; i < NGPU; i++) {
		hipSetDevice(i);
		err = hipStreamCreate(&s[i]);
	}
}

/*
void Benchmark22::init(){
	for(int i = 0; i<N; i++){
		for(int j = 0; j<N; j++){
			U[i][j] = 0.0;
			LT[i][j] = 0.0;
			A[i][j] = float(rand()%100 + 1);
			if(rand()%2)
				A[i][j] *= -1;
			A_hw(i,j,N) = A[i][j];
			U_hw(i,j,N) = 0.0;
			L_hw(i,j,N) = 0.0;
		}
	}
}

void Benchmark22::reset(){
	srand(time(0));
	for(int i = 0; i<N; i++){
		for(int j = 0; j<N; j++){
			U[i][j] = 0.0;
			LT[i][j] = 0.0;
			A[i][j] = float(rand()%100 + 1);
			if(rand()%2)
				A[i][j] *= -1;
			A_hw(i,j,N) = A[i][j];
			U_hw(i,j,N) = 0.0;
			L_hw(i,j,N) = 0.0;
		}
	}
}

void Benchmark22::execute_sync(int iter){
	
	printf("\nExecuting SW version (transpose)");
	for (int it = 0; it < N; it++){
		SWupdateUT(A, U, LT, N, it);
		SWupdateLT(A, U, LT, N, it);
	}
	
	printf("\nExecuting on GPU\n");
	hipSetDevice(0);
	for (int it = 0; it < N; it++){
		updateU<<<num_blocks,block_size_1d>>>(A, U, L, N, it);
		hipDeviceSynchronize();
		updateL<<<num_blocks,block_size_1d>>>(A, U, L, N, it);
		hipDeviceSynchronize();
	}
}
*/

void Benchmark22::init(){
	for(int i = 0; i<N; i++){
		for(int j = 0; j<N; j++){
			U(i,j,N) = 0.0;
			L(i,j,N) = 0.0;
			A(i,j,N) = float(rand()%100 + 1);
			if(rand()%2)
				A(i,j,N) *= -1;
		}
	}
}

void Benchmark22::reset(){
	srand(time(0));
	for(int i = 0; i<N; i++){
		for(int j = 0; j<N; j++){
			U(i,j,N) = 0.0;
			L(i,j,N) = 0.0;
			A(i,j,N) = float(rand()%100 + 1);
			if(rand()%2)
				A(i,j,N) *= -1;
		}
	}
}

void Benchmark22::execute_sync(int iter){
	int ngpu = NGPU; // NGPU;
	for (int it = 0; it < N; it++){
		for(int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			updateU<<<(N/block_size_1d+1)/ngpu+1,block_size_1d>>>(A, U, L, g*(N/ngpu+1),(g+1)*(N/ngpu+1), N, it);
			hipDeviceSynchronize();
		}
		for(int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			updateL<<<(N/block_size_1d+1)/ngpu+1,block_size_1d>>>(A, U, L, g*(N/ngpu+1),(g+1)*(N/ngpu+1), N, it);
			hipDeviceSynchronize();
		}
	}
}

void Benchmark22::execute_async(int iter){

	// if (!pascalGpu || stream_attach) {
	// 	for (int i = 0; i < NGPU; i++) {
	// 		hipStreamAttachMemAsync(s[i], A, long(N)*long(N)*sizeof(float));
	// 		hipStreamAttachMemAsync(s[i], L, long(N)*long(N)*sizeof(float));
	// 		hipStreamAttachMemAsync(s[i], U, long(N)*long(N)*sizeof(float));
	// 	}
    // }

	// if (pascalGpu && do_prefetch) {
	// 	for (int i = 0; i < NGPU; i++) {
	// 		hipMemPrefetchAsync(A, long(N)*long(N)*sizeof(float), i, s[i]);
	// 		hipMemPrefetchAsync(L, long(N)*long(N)*sizeof(float), i, s[i]);
	// 		hipMemPrefetchAsync(U, long(N)*long(N)*sizeof(float), i, s[i]);
	// 	}
    // }

	hipEvent_t e1, e2;
	int og_slice = N/NGPU+1;

	for (int it = 0; it < N; it++){
		int slice = (N-it)/NGPU+1;

		for(int g = 0; g < NGPU; g++){		
			hipSetDevice(g);

			// v1
			updateU<<<(N/block_size_1d+1)/NGPU+1,block_size_1d,0,s[g]>>>(A, U, L, (it>g*og_slice)?it:g*og_slice,(it>(g+1)*og_slice)?it:(g+1)*og_slice, N, it);
			if(it/og_slice == g){ 
				hipEventCreate(&e1);
				hipEventRecord(e1, s[g]);
			}

			// v2
			// int offset = (it/NGPU)*NGPU+g*slice;
			// int limit = (it/NGPU)*NGPU+(g+1)*slice;
			// updateU<<<(N/block_size_1d+1)/NGPU+1,block_size_1d,0,s[g]>>>(A, U, L, (it>offset)?it:offset, (it>limit)?it:limit, N, it);
			// if(g == 0){ 
			// 	hipEventCreate(&e1);
			// 	hipEventRecord(e1, s[0]);
			// }

		}

		for(int g = 0; g < NGPU; g++){
			hipStreamWaitEvent(s[g], e1, 0);
		}

        for(int g = 0; g < NGPU; g++){		
			hipSetDevice(g);

			// v1
			updateL<<<(N/block_size_1d+1)/NGPU+1,block_size_1d,0,s[g]>>>(A, U, L, (it>g*og_slice)?it:g*og_slice,(it>(g+1)*og_slice)?it:(g+1)*og_slice, N, it);
			if(it/og_slice == g){ 
				hipEventCreate(&e2);
				hipEventRecord(e2, s[g]);
			}

			// v2
			// int offset = (it/NGPU)*NGPU+g*slice;
			// int limit = (it/NGPU)*NGPU+(g+1)*slice;
			// updateL<<<(N/block_size_1d+1)/NGPU+1,block_size_1d,0,s[g]>>>(A, U, L, (it>offset)?it:offset, (it>limit)?it:limit, N, it);
			// if(g == 0){ 
			// 	hipEventCreate(&e2);
			// 	hipEventRecord(e2, s[0]);
			// }
		}

		for(int g = 0; g < NGPU; g++){
			hipStreamWaitEvent(s[g], e2, 0);
		}
	}

	for (int j = 0; j < NGPU; j++) {
		hipSetDevice(j);
		hipDeviceSynchronize();
		// err = hipStreamSynchronize(s[j]);
	}

}

void Benchmark22::execute_cudagraph(int iter){}
void Benchmark22::execute_cudagraph_manual(int iter){}
void Benchmark22::execute_cudagraph_single(int iter){
	int ngpu = 1;
	for (int it = 0; it < N; it++){
		for(int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			updateU<<<(N/block_size_1d+1)/ngpu+1,block_size_1d>>>(A, U, L, g*(N/ngpu+1),(g+1)*(N/ngpu+1), N, it);
			hipDeviceSynchronize();
		}
		for(int g = 0; g < ngpu; g++){
			hipSetDevice(g);
			updateL<<<(N/block_size_1d+1)/ngpu+1,block_size_1d>>>(A, U, L, g*(N/ngpu+1),(g+1)*(N/ngpu+1), N, it);
			hipDeviceSynchronize();
		}
	}
}

std::string Benchmark22::print_result(bool short_form){
	/*
	
	float tolerance = 0.0001;
	for(int i = 0; i<N; i++){
		for(int j = 0; j<N; j++){
			if(abs(U[i][j]-U_hw(i,j,N)) >= tolerance){
				tolerance = abs(U[i][j]-U_hw(i,j,N));
				// printf("(%d,%d) \t U= %f vs %f\terror = %f\n", i,j,U[i][j],U_hw(i,j,N),abs(U[i][j]-U_hw(i,j,N)));
			}
			if(abs(LT[i][j]-L_hw(i,j,N)) >= tolerance){
				tolerance = abs(LT[i][j]-L_hw(i,j,N));
				// printf("(%d,%d) \t LT= %f vs %f\terror = %f\n", i,j,LT[i][j],L_hw(i,j,N),abs(LT[i][j]-L_hw(i,j,N)));
			}
		}
	}
	printf("\n\n");
	*/
	std::string res;
	float err = 0.0;
	for(int i = N-50; i<N; i++)
		err += abs(1.0-L(i,i,N));
	return std::to_string(err);    
} 
