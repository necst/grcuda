#include "hip/hip_runtime.h"
#include "b11.cuh"
#include <thread>
#include <vector>
//////////////////////////////
//////////////////////////////

__global__ void squareMulti(const float *x, float *y, int n)
{
    for (int i  = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        // float tmp = x[i];
        // float sum = 0;
        // for (int j = 0; j < 4; j++) {
        //     sum += tmp + j;
        // }

        y[i] = x[i] * x[i]; // tmp + tmp * tmp / 2 + tmp * tmp * tmp / 6;
    }
}

__inline__ __device__ float warp_reduceMulti(float val)
{
    int warp_size = 32;
    for (int offset = warp_size / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

// __device__ float atomicAddDouble(float* address, float val) {
//     unsigned long long int* address_as_ull = (unsigned long long int*) address;
//     unsigned long long int old = *address_as_ull, assumed;
//     do {
//         assumed = old;
//         old = atomicCAS(address_as_ull, assumed, __float_as_longlong(val + __longlong_as_float(assumed)));
//     } while (assumed != old);
//     return __longlong_as_float(old);
// }

__global__ void reduceMulti(const float *x, const float *y, float *z, int N)
{
    int warp_size = 32;
    float sum = float(0);
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        sum += x[i] - y[i];
    }
    sum = warp_reduceMulti(sum);                   // Obtain the sum of values in the current warp;
    if ((threadIdx.x & (warp_size - 1)) == 0) // Same as (threadIdx.x % warp_size) == 0 but faster
        atomicAdd(z, sum);                    // The first thread in the warp updates the output;
}

//////////////////////////////
//////////////////////////////

void Benchmark11::alloc()
{
    hipSetDevice(0);            // Set device 0 as current
    err = hipMallocManaged(&x, sizeof(float) * N);
    err = hipMallocManaged(&x1, sizeof(float) * N);
    err = hipStreamCreate(&s1);
    hipSetDevice(1);            // Set device 1 as current
    err = hipMallocManaged(&y, sizeof(float) * N);
    err = hipMallocManaged(&y1, sizeof(float) * N);
    err = hipMallocManaged(&res, sizeof(float));
    err = hipStreamCreate(&s2);
}

void Benchmark11::init()
{
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0 / (i + 1);
        y[i] = 2.0 / (i + 1);
    }
}

void Benchmark11::reset()
{
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0 / (i + 1);
        y[i] = 2.0 / (i + 1);
    }
    res[0] = 0.0;
}

void Benchmark11::execute_sync(int iter)
{
    squareMulti<<<num_blocks, block_size_1d>>>(x, x1, N);
    err = hipDeviceSynchronize();
    squareMulti<<<num_blocks, block_size_1d>>>(y, y1, N);
    err = hipDeviceSynchronize();
    reduceMulti<<<num_blocks, block_size_1d>>>(x1, y1, res, N);
    err = hipDeviceSynchronize();
}


void Benchmark11::execute_async(int iter)
{


    if (pascalGpu && do_prefetch) {
        hipMemPrefetchAsync(x, sizeof(float) * N, 0, s1);
        hipMemPrefetchAsync(x1, sizeof(float) * N, 0, s1);
        hipMemPrefetchAsync(y, sizeof(float) * N, 1, s2);
        hipMemPrefetchAsync(y1, sizeof(float) * N, 1, s2);
        hipMemPrefetchAsync(res, sizeof(float), 0, s1);
    }

    hipSetDevice(0);            // Set device 0 as current


    hipStreamAttachMemAsync(s1, x, sizeof(float) * N);
    hipStreamAttachMemAsync(s1, x1, sizeof(float) * N);
    squareMulti<<<num_blocks, block_size_1d, 0, s1>>>(x, x1, N);

    hipSetDevice(1);            // Set device 1 as current
    hipStreamAttachMemAsync(s2, y, sizeof(float) * N);
    hipStreamAttachMemAsync(s2, y1, sizeof(float) * N);
    squareMulti<<<num_blocks, block_size_1d, 0, s2>>>(y, y1, N);

    // Stream 1 waits stream 2;
    hipEvent_t e1;
    hipEventCreate(&e1);
    hipEventRecord(e1, s2);
    hipStreamWaitEvent(s1, e1, 0);
    hipSetDevice(0);

    
    hipStreamAttachMemAsync(s1, y1, sizeof(float) * N);

    if (pascalGpu && do_prefetch) {
        hipMemPrefetchAsync(y1, sizeof(float) * N, 0, s1);
    }
    reduceMulti<<<num_blocks, block_size_1d, 0, s1>>>(x1, y1, res, N);
    hipStreamSynchronize(s1);




}





void Benchmark11::execute_cudagraph(int iter) {}

void Benchmark11::execute_cudagraph_manual(int iter) {}

void Benchmark11::execute_cudagraph_single(int iter) {}

std::string Benchmark11::print_result(bool short_form)
{
    return std::to_string(res[0]);
}
