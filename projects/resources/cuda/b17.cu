#include "hip/hip_runtime.h"
#include "b17.cuh"

//////////////////////////////
//////////////////////////////

#define WARP_SIZE 32
#define THREADS_PER_VECTOR 4
#define MAX_NUM_VECTORS_PER_BLOCK (1024 / THREADS_PER_VECTOR)

/////////////////////////////
/////////////////////////////

extern "C" __global__ void spmv_multi(const int *ptr, const int *idx, const int *val, const float *vec, float *res, int num_rows, int num_nnz) {
    for (int n = blockIdx.x * blockDim.x + threadIdx.x; n < num_rows; n += blockDim.x * gridDim.x) {
        float sum = 0;
        for (int i = ptr[n]; i < ptr[n + 1]; i++) {
            sum += val[i] * vec[idx[i]];
        }
        res[n] = sum;
    }
}

extern "C" __global__ void spmv2_multi(const int *ptr, const int *idx, const int *val, const float *vec, float *res, int num_rows, int num_nnz) {
    // Thread ID in block
    int t = threadIdx.x;

    // Thread ID in warp
    int lane = t & (WARP_SIZE - 1);

    // Number of warps per block
    int warpsPerBlock = blockDim.x / WARP_SIZE;

    // One row per warp
    int row = (blockIdx.x * warpsPerBlock) + (t / WARP_SIZE);

    extern __shared__ volatile float vals[];

    if (row < num_rows) {
        int rowStart = ptr[row];
        int rowEnd = ptr[row + 1];
        float sum = 0;

        // Use all threads in a warp accumulate multiplied elements
        for (int j = rowStart + lane; j < rowEnd; j += WARP_SIZE) {
            int col = idx[j];
            sum += val[j] * vec[col];
        }
        vals[t] = sum;
        __syncthreads();

        // Reduce partial sums
        if (lane < 16) vals[t] += vals[t + 16];
        if (lane < 8) vals[t] += vals[t + 8];
        if (lane < 4) vals[t] += vals[t + 4];
        if (lane < 2) vals[t] += vals[t + 2];
        if (lane < 1) vals[t] += vals[t + 1];
        __syncthreads();

        // Write result
        if (lane == 0) {
            res[row] = vals[t];
        }
    }
}

extern "C" __global__ void spmv3_multi(int *cudaRowCounter, int *d_ptr, int *d_cols, int *d_val, float *d_vector, float *d_out, int N) {
    int i;
    float sum;
    int row;
    int rowStart, rowEnd;
    int laneId = threadIdx.x % THREADS_PER_VECTOR;       //lane index in the vector
    int vectorId = threadIdx.x / THREADS_PER_VECTOR;     //vector index in the thread block
    int warpLaneId = threadIdx.x & 31;                   //lane index in the warp
    int warpVectorId = warpLaneId / THREADS_PER_VECTOR;  //vector index in the warp

    __shared__ volatile int space[MAX_NUM_VECTORS_PER_BLOCK][2];

    // Get the row index
    if (warpLaneId == 0) {
        row = atomicAdd(cudaRowCounter, 32 / THREADS_PER_VECTOR);
    }
    // Broadcast the value to other threads in the same warp and compute the row index of each vector
    row = __shfl_sync(0xffffffff, row, 0) + warpVectorId;

    while (row < N) {
        // Use two threads to fetch the row offset
        if (laneId < 2) {
            space[vectorId][laneId] = d_ptr[row + laneId];
        }
        rowStart = space[vectorId][0];
        rowEnd = space[vectorId][1];

        sum = 0;
        // Compute dot product
        if (THREADS_PER_VECTOR == 32) {
            // Ensure aligned memory access
            i = rowStart - (rowStart & (THREADS_PER_VECTOR - 1)) + laneId;

            // Process the unaligned part
            if (i >= rowStart && i < rowEnd) {
                sum += d_val[i] * d_vector[d_cols[i]];
            }

            // Process the aligned part
            for (i += THREADS_PER_VECTOR; i < rowEnd; i += THREADS_PER_VECTOR) {
                sum += d_val[i] * d_vector[d_cols[i]];
            }
        } else {
            for (i = rowStart + laneId; i < rowEnd; i += THREADS_PER_VECTOR) {
                sum += d_val[i] * d_vector[d_cols[i]];
            }
        }
        // Intra-vector reduction
        for (i = THREADS_PER_VECTOR >> 1; i > 0; i >>= 1) {
            sum += __shfl_down_sync(0xffffffff, sum, i);
        }

        // Save the results
        if (laneId == 0) {
            d_out[row] = sum;
        }

        // Get a new row index
        if (warpLaneId == 0) {
            row = atomicAdd(cudaRowCounter, 32 / THREADS_PER_VECTOR);
        }
        // Broadcast the row index to the other threads in the same warp and compute the row index of each vector
        row = __shfl_sync(0xffffffff, row, 0) + warpVectorId;
    }
}

__inline__ __device__ float warp_reduce_multi(float val) {
    int warp_size = 32;
    for (int offset = warp_size / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

extern "C" __global__ void sum_multi(const float *x, float *z, int N) {
    int warp_size = 32;
    float sum = float(0);
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        sum += x[i];
    }
    sum = warp_reduce_multi(sum);                    // Obtain the sum of values in the current warp;
    if ((threadIdx.x & (warp_size - 1)) == 0)  // Same as (threadIdx.x % warp_size) == 0 but faster
        atomicAdd(z, sum);                     // The first thread in the warp updates the output;
}

extern "C" __global__ void divide_multi(const float *x, float *y, float *val, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        y[i] = x[i] / val[0];
    }
}

//////////////////////////////
//////////////////////////////

void Benchmark17::alloc() {
    nnz = degree * N;
    ptr_tmp = (int *)malloc(sizeof(int) * (N + 1));
    ptr2_tmp = (int *)malloc(sizeof(int) * (N + 1));
    idx_tmp = (int *)malloc(sizeof(int) * nnz);
    idx2_tmp = (int *)malloc(sizeof(int) * nnz);
    val_tmp = (int *)malloc(sizeof(int) * nnz);
    val2_tmp = (int *)malloc(sizeof(int) * nnz);

    hipSetDevice(0);
    err = hipMallocManaged(&ptr2, sizeof(int) * (N + 1));
    err = hipMallocManaged(&idx2, sizeof(int) * nnz);
    err = hipMallocManaged(&val2, sizeof(int) * nnz);
    err = hipMallocManaged(&hub1, sizeof(float) * N);
    err = hipMallocManaged(&auth2, sizeof(float) * N);
    err = hipMallocManaged(&rowCounter1, sizeof(int));
    err = hipMallocManaged(&auth_norm, sizeof(float));
    err = hipStreamCreate(&s1);

    hipSetDevice(1);
    err = hipMallocManaged(&ptr, sizeof(int) * (N + 1));
    err = hipMallocManaged(&idx, sizeof(int) * nnz);
    err = hipMallocManaged(&val, sizeof(int) * nnz);
    err = hipMallocManaged(&auth1, sizeof(float) * N);
    err = hipMallocManaged(&hub2, sizeof(float) * N);
    err = hipMallocManaged(&rowCounter2, sizeof(int));
    err = hipMallocManaged(&hub_norm, sizeof(float));
    err = hipStreamCreate(&s2);


    x = (int *)malloc(nnz * sizeof(int));
    y = (int *)malloc(nnz * sizeof(int));
    v = (int *)malloc(nnz * sizeof(int));

}

void Benchmark17::init() {
    random_coo(x, y, v, N, degree);
    // Create a CSR;
    coo2csr(ptr_tmp, idx_tmp, val_tmp, x, y, v, N, N, nnz);
    coo2csr(ptr2_tmp, idx2_tmp, val2_tmp, y, x, v, N, N, nnz);
}

void Benchmark17::reset() {
    hipSetDevice(0);

    for (int j = 0; j < nnz; j++) {
        idx2[j] = idx2_tmp[j];
        val2[j] = val2_tmp[j];
    }
    for (int j = 0; j < N + 1; j++) {
        ptr2[j] = ptr2_tmp[j];
    }
    for (int i = 0; i < N; i++) {
        auth2[i] = 1;
        hub1[i] = 1;
    }
    auth_norm[0] = 0;
    rowCounter1[0] = 0;
    hipSetDevice(1);

    for (int j = 0; j < nnz; j++) {
        idx[j] = idx_tmp[j];
        val[j] = val_tmp[j];
    }
    for (int j = 0; j < N + 1; j++) {
        ptr[j] = ptr_tmp[j];
    }
    for (int i = 0; i < N; i++) {
        auth1[i] = 1;
        hub2[i] = 1;
    }
    hub_norm[0] = 0;
    rowCounter2[0] = 0;
}

void Benchmark17::execute_sync(int iter) {
    for (int iter = 0; iter < iterations; iter++) {
        // hipMemPrefetchAsync(auth1, N * sizeof(float), 0);
        // hipMemPrefetchAsync(auth2, N * sizeof(float), 0);
        // hipMemPrefetchAsync(hub1, N * sizeof(float), 0);
        // hipMemPrefetchAsync(hub2, N * sizeof(float), 0);
        // hipMemPrefetchAsync(auth_norm, sizeof(float), 0);
        // hipMemPrefetchAsync(hub_norm, sizeof(float), 0);
        // hipDeviceSynchronize();

        int nb = ceil(N / ((float)block_size_1d));

        // spmv<<<nb, block_size_1d>>>(ptr2, idx2, val2, hub1, auth2, N, nnz);
        spmv3_multi<<<nb, block_size_1d, block_size_1d * sizeof(float)>>>(rowCounter1, ptr2, idx2, val2, hub1, auth2, N);
        err = hipDeviceSynchronize();

        // spmv<<<nb, block_size_1d>>>(ptr, idx, val, auth1, hub2, N, nnz);
        spmv3_multi<<<nb, block_size_1d, block_size_1d * sizeof(float)>>>(rowCounter2, ptr, idx, val, auth1, hub2, N);
        err = hipDeviceSynchronize();

        sum_multi<<<num_blocks, block_size_1d>>>(auth2, auth_norm, N);
        err = hipDeviceSynchronize();

        sum_multi<<<num_blocks, block_size_1d>>>(hub2, hub_norm, N);
        err = hipDeviceSynchronize();

        divide_multi<<<num_blocks, block_size_1d>>>(auth2, auth1, auth_norm, N);
        err = hipDeviceSynchronize();

        divide_multi<<<num_blocks, block_size_1d>>>(hub2, hub1, hub_norm, N);
        err = hipDeviceSynchronize();

        auth_norm[0] = 0;
        hub_norm[0] = 0;
        rowCounter1[0] = 0;
        rowCounter2[0] = 0;

        if (debug && err) std::cout << err << std::endl;
    }
}

void Benchmark17::execute_async(int iter) {
    for (int iter = 0; iter < iterations; iter++) {
        // hipMemPrefetchAsync(auth1, N * sizeof(float), 0, s2);
        // hipMemPrefetchAsync(auth2, N * sizeof(float), 0, s1);
        // hipMemPrefetchAsync(hub1, N * sizeof(float), 0, s1);
        // hipMemPrefetchAsync(hub2, N * sizeof(float), 0, s2);
        // hipMemPrefetchAsync(auth_norm, sizeof(float), 0, s1);
        // hipMemPrefetchAsync(hub_norm, sizeof(float), 0, s2);
        hipSetDevice(0);
        hipStreamAttachMemAsync(s1, ptr2, 0);
        hipStreamAttachMemAsync(s1, idx2, 0);
        hipStreamAttachMemAsync(s1, val2, 0);
        hipStreamAttachMemAsync(s1, hub1, 0);
        hipStreamAttachMemAsync(s1, auth2, 0);
        hipStreamAttachMemAsync(s1, rowCounter1, 0);
        hipStreamAttachMemAsync(s1, auth_norm, 0);
        hipEvent_t e1;
        hipEventCreate(&e1);

        hipSetDevice(1);
        hipStreamAttachMemAsync(s2, ptr, 0);
        hipStreamAttachMemAsync(s2, idx, 0);
        hipStreamAttachMemAsync(s2, val, 0);
        hipStreamAttachMemAsync(s2, auth1, 0);
        hipStreamAttachMemAsync(s2, hub2, 0);
        hipStreamAttachMemAsync(s2, rowCounter2, 0);
        hipStreamAttachMemAsync(s2, hub_norm, 0);
        hipEvent_t e2;
        hipEventCreate(&e2);


        if (pascalGpu && do_prefetch) {
            hipMemPrefetchAsync(auth1, N * sizeof(float), 1, s2);
            hipMemPrefetchAsync(auth2, N * sizeof(float), 0, s1);
            hipMemPrefetchAsync(hub1, N * sizeof(float), 0, s1);
            hipMemPrefetchAsync(hub2, N * sizeof(float), 1, s2);
            hipMemPrefetchAsync(auth_norm, sizeof(float), 0, s1);
            hipMemPrefetchAsync(hub_norm, sizeof(float), 1, s2);
        }
        int nb = ceil(N / ((float)block_size_1d));
        hipSetDevice(0);
        // spmv<<<nb, block_size_1d, 0, s1>>>(ptr2, idx2, val2, hub1, auth2, N, nnz);
        spmv3_multi<<<nb, block_size_1d, block_size_1d * sizeof(float), s1>>>(rowCounter1, ptr2, idx2, val2, hub1, auth2, N);
        err = hipEventRecord(e1, s1);

        hipSetDevice(1);
        // spmv<<<nb, block_size_1d, 0, s2>>>(ptr, idx, val, auth1, hub2, N, nnz);
        spmv3_multi<<<nb, block_size_1d, block_size_1d * sizeof(float), s2>>>(rowCounter2, ptr, idx, val, auth1, hub2, N);
        err = hipEventRecord(e2, s2);

        hipSetDevice(0);
        sum_multi<<<num_blocks, block_size_1d, 0, s1>>>(auth2, auth_norm, N);
        hipSetDevice(1);
        sum_multi<<<num_blocks, block_size_1d, 0, s2>>>(hub2, hub_norm, N);

        // Stream 1 waits stream 2;
        hipSetDevice(0);
        err = hipStreamWaitEvent(s1, e2, 0);
        hipStreamAttachMemAsync(s1, auth1, 0);

        if (pascalGpu && do_prefetch) {
            hipMemPrefetchAsync(auth1, N * sizeof(float), 1, s1);
        }
        divide_multi<<<num_blocks, block_size_1d, 0, s1>>>(auth2, auth1, auth_norm, N);
        // Stream 2 waits stream 1;
        hipSetDevice(1);
        err = hipStreamWaitEvent(s2, e1, 0);
        hipStreamAttachMemAsync(s2, hub1, 0);
        if (pascalGpu && do_prefetch) {
            hipMemPrefetchAsync(hub1, N * sizeof(float), 1, s2);
        }
        divide_multi<<<num_blocks, block_size_1d, 0, s2>>>(hub2, hub1, hub_norm, N);

        hipSetDevice(0);
        err = hipStreamSynchronize(s1);
        auth_norm[0] = 0;
        rowCounter1[0] = 0;

        hipSetDevice(1);
        err = hipStreamSynchronize(s2);
        hub_norm[0] = 0;
        rowCounter2[0] = 0;

        if (debug && err) std::cout << err << std::endl;
    }
}

void Benchmark17::execute_cudagraph(int iter) {}

void Benchmark17::execute_cudagraph_manual(int iter) {}

void Benchmark17::execute_cudagraph_single(int iter) {}
std::string Benchmark17::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(auth1[0]);
    } else {
        std::string res = "[";
        for (int j = 0; j < 10; j++) {
            res += std::to_string(auth1[j]) + ", ";
        }
        return res + ", ...]";
    }
}