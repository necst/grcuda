#include "hip/hip_runtime.h"
// Copyright (c) 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.


#include "b12.cuh"

/**
 * From https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
 */
#define CUDA_CHECK_ERROR(kernel_ret_code) { gpuAssert((kernel_ret_code), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
    if (code != hipSuccess){
    }
}

__global__ void subtract(float* v1, const float* v2, const float alpha, int N, int offset) {
    int init = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = init; i < N; i += stride){
        v1[i] -= alpha * v2[i + offset];
    }
}

__global__ void copy_partition_to_vec(const float *vec_in, float *vec_out, const int N, const int offset_out, const int offset_in){
    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = init; i < N; i += stride){
        vec_out[i + offset_out] = vec_in[i + offset_in];
    }
}

__global__ void normalize(const float *d_v_in, const float denominator, float *d_v_out, int N) {
    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < N; i += stride) {
        d_v_out[i] = d_v_in[i] * denominator;
    }
}

__global__ void spmv(const int *x, const int *y, const float *val, const float *v_in, float *v_out, int num_nnz) {
    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    //printf("BlockID: %d, ThreadID: %d. Init = %d, Stride = %d\n", blockIdx.x, threadIdx.x, init, stride);

    for (int i = init; i < num_nnz; i += stride) {
        //printf("v_out[%d] += v_in[%d] * val[%d]\n", y[i], x[i], i);
        v_out[y[i]] += v_in[x[i]] * val[i];
    }
}

__global__ void dot_product_stage_one(const float* v1, const float* v2, float* temporaryOutputValues, int N, int offset) {
    extern __shared__ float cache[];
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIdx = threadIdx.x;

    int stride = blockDim.x * gridDim.x;
    float temp = 0;
    for(int i = threadId; i < N; i += stride){
        temp += v1[i] * v2[i + offset];
    }

    cache[cacheIdx] = temp;

    __syncthreads();

    for(int i = blockDim.x >> 1; i != 0; i >>= 1){
        if(cacheIdx < i){
            cache[cacheIdx] += cache[cacheIdx + 1];
        }
        __syncthreads();
    }

    if (cacheIdx == 0){
        temporaryOutputValues[blockIdx.x] = cache[0];
    }
}

__global__ void dot_product_stage_two(const float *temporary_results, float *result) {

    float acc = temporary_results[threadIdx.x];
    for(int i = 16; i > 0; i >>= 1){
        acc += __shfl_down_sync(0xffffffff, acc, i);
        __syncthreads();
    }

    __syncthreads();

    if(threadIdx.x == 0) *result = acc;
}

__global__ void axpb_xtended(const float alpha, const float *x, const float *b, const float beta, const float *c, float *out, const int N, const int offset_x, const int offset_c) {
    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = init; i < N; i += stride) {
        out[i] = alpha * x[i + offset_x] + b[i] + beta * c[i + offset_c];
    }
}


void Benchmark12::alloc_vectors() {
    for (const auto& partition: this->coo_partitions){
        f32 *tmp_vec_in, *tmp_spmv_out, *tmp_intermediate_dot_product_values;
        f32 *tmp_vec_next, *tmp_lanczos_vectors, *tmp_normalized_out;

        CUDA_CHECK_ERROR(hipMallocManaged(&tmp_vec_in, sizeof(f32) * this->matrix.N));
        CUDA_CHECK_ERROR(hipMallocManaged(&tmp_spmv_out, sizeof(f32) * partition->N));
        CUDA_CHECK_ERROR(hipMallocManaged(&tmp_intermediate_dot_product_values, sizeof(f32) * 32));
        CUDA_CHECK_ERROR(hipMallocManaged(&tmp_vec_next, sizeof(f32) * partition->N));
        CUDA_CHECK_ERROR(hipMallocManaged(&tmp_lanczos_vectors, sizeof(f32) * this->num_eigencomponents * partition->N));
        CUDA_CHECK_ERROR(hipMallocManaged(&tmp_normalized_out, sizeof(f32) * partition->N));

        this->vec_in.push_back(tmp_vec_in);
        this->spmv_vec_out.push_back(tmp_spmv_out);
        this->intermediate_dot_product_values.push_back(tmp_intermediate_dot_product_values);
        this->vec_next.push_back(tmp_vec_next);
        this->lanczos_vectors.push_back(tmp_lanczos_vectors);
        this->normalized_out.push_back(tmp_normalized_out);
    }

    CUDA_CHECK_ERROR(hipMallocManaged(&alpha_intermediate, sizeof(f32) * this->num_gpus));
    CUDA_CHECK_ERROR(hipMallocManaged(&beta_intermediate, sizeof(f32) * this->num_gpus));
}

void Benchmark12::alloc_coo_partitions() {

    const u32 nnz_per_partition = u32((this->matrix.nnz + this->num_gpus) / this->num_gpus);
    u32 from_index = 0;
    u32 to_index = nnz_per_partition;
    u32 index_value = this->matrix.y[to_index];

    for(u32 i = 0; i < this->num_gpus - 1; ++i){
        while(index_value == this->matrix.y[to_index]) {
            to_index++;
        }
        const u32 offset = (from_index == 0) ? from_index : (this->matrix.y[from_index] - 1);
        auto coo_partition = (this->assign_partition(from_index, to_index, offset));
        this->coo_partitions.push_back(coo_partition);

        from_index = to_index;
        to_index += nnz_per_partition;
        index_value = this->matrix.y[to_index];
    }
    const u32 offset = this->matrix.y[from_index];
    auto coo_partition = (this->assign_partition(from_index, this->matrix.nnz, offset));
    this->coo_partitions.push_back(coo_partition);
}

coo_matrix_t *Benchmark12::assign_partition(u32 from_index, u32 to_index, u32 offset) {
    i32 *tmp_x, *tmp_y;
    f32 *tmp_val;
    coo_matrix_t *coo_partition;
    hipMallocManaged(&coo_partition, sizeof(coo_matrix_t));
    coo_partition->begin = from_index;
    coo_partition->end = to_index;
    CUDA_CHECK_ERROR(hipMallocManaged(&tmp_x, sizeof(u32) * (to_index - from_index)));
    CUDA_CHECK_ERROR(hipMallocManaged(&tmp_y, sizeof(u32) * (to_index - from_index)));
    CUDA_CHECK_ERROR(hipMallocManaged(&tmp_val, sizeof(f32) * (to_index - from_index)));

    coo_partition->x = tmp_x;
    coo_partition->y = tmp_y;
    coo_partition->val = tmp_val;

    u32 j = 0;
    for(u32 i = from_index; i < to_index; ++i,++j){
        coo_partition->x[j]   = this->matrix.x[i];
        coo_partition->y[j]   = this->matrix.y[i] - offset;
        coo_partition->val[j] = this->matrix.val[i];
    }

    coo_partition->N = coo_partition->y[to_index - from_index - 1] + 1;
    coo_partition->nnz = to_index - from_index;
    return coo_partition;
}

void Benchmark12::create_random_matrix(bool normalize = true) {
    u32 total_nnz = RANDOM_MATRIX_AVG_NNZ_PER_ROW * RANDOM_MATRIX_NUM_ROWS;
    i32 *x        = (i32*) std::malloc(total_nnz * sizeof(i32));
    i32 *y        = (i32*) std::malloc(total_nnz * sizeof(i32));
    f32 *val      = (f32*) std::malloc(total_nnz * sizeof(f32));

    f32 value_to_set = normalize ? (1.0f / RANDOM_MATRIX_NUM_ROWS) : 1.0f;

    for(u32 i = 0; i < total_nnz; ++i)
        val[i] = value_to_set;

    auto random_node = [&](){
        return std::rand() % RANDOM_MATRIX_NUM_ROWS;
    };


    std::generate(x, x + total_nnz, random_node);
    std::generate(y, y + total_nnz, random_node);

    std::sort(y, y + total_nnz);

    this->matrix.x = x;
    this->matrix.y = y;
    this->matrix.val = val;
    this->matrix.begin = 0;
    this->matrix.end = total_nnz;
    this->matrix.N = RANDOM_MATRIX_NUM_ROWS;
    this->matrix.nnz = total_nnz;

}

void Benchmark12::alloc() {

    if(this->matrix_path.empty())
        this->create_random_matrix();

    this->create_streams();
    this->alloc_coo_partitions();
    this->alloc_vectors();

    // Create offsets
    this->offsets.push_back(0);
    for(u32 i = 1; i < this->num_gpus; ++i)
        this->offsets.push_back(this->coo_partitions[i]->N - this->offsets[i - 1]);

}

void Benchmark12::reset() {
    // Just call init, it resets all the necessary vectors;
    this->init();
}

void Benchmark12::create_streams() {

    for(u32 i = 0; i < this->num_gpus; ++i){
        hipSetDevice(i);
        hipStream_t *stream = (hipStream_t*) std::malloc(sizeof(hipStream_t));
        hipStreamCreate(stream);
        this->streams[i] = *stream;
    }

    hipSetDevice(0);

}

template <typename Function>
void Benchmark12::launch_multi_kernel(Function kernel_launch_function) {


    for(u32 i = 0; i < this->num_gpus; ++i) {
        CUDA_CHECK_ERROR(hipSetDevice(i));
        hipStream_t stream = policy == Policy::Sync ? nullptr : streams[i];
        kernel_launch_function(i, stream);

        if(policy == Policy::Sync)
            hipDeviceSynchronize();
    }

}

void Benchmark12::execute(i32 iter) {


    f32 alpha = 0.0f;
    f32 beta  = 0.0f;
    f32* alpha_storage_host = (f32 *) std::malloc(this->num_gpus * sizeof(f32));
    f32* beta_storage_host = (f32 *) std::malloc(this->num_gpus * sizeof(f32));

    if(this->debug)
        std::cout << "[LANCZOS - Sync] Iteration " << iter << std::endl;



    this->launch_multi_kernel([&](u32 p_idx, hipStream_t stream){

        spmv<<<this->num_blocks, this->block_size, 0, stream>>>(
                this->coo_partitions[p_idx]->x,
                this->coo_partitions[p_idx]->y,
                this->coo_partitions[p_idx]->val,
                this->vec_in[p_idx],
                this->spmv_vec_out[p_idx],
                this->coo_partitions[p_idx]->nnz
        );
    });

    this->launch_multi_kernel([&](u32 p_idx, hipStream_t stream){
        dot_product_stage_one<<<DOT_PRODUCT_NUM_BLOCKS, this->block_size * (this->num_blocks / DOT_PRODUCT_NUM_BLOCKS), 4 * this->block_size * (this->num_blocks / DOT_PRODUCT_NUM_BLOCKS), stream>>>(
            this->vec_in[p_idx],
            this->spmv_vec_out[p_idx],
            this->intermediate_dot_product_values[p_idx],
            this->coo_partitions[p_idx]->N,
            this->offsets[p_idx]
        );
        dot_product_stage_two<<<1, 32, 0, stream>>>(
            this->intermediate_dot_product_values[p_idx],
            &this->alpha_intermediate[p_idx]
        );
    });

    hipDeviceSynchronize();

    hipMemcpy(alpha_storage_host, this->alpha_intermediate, this->num_gpus, hipMemcpyDeviceToHost);
    alpha = std::accumulate(alpha_storage_host, alpha_storage_host + this->num_gpus, 0.0f);
    tridiagonal_matrix.push_back(alpha);

    this->launch_multi_kernel([&, alpha](u32 p_idx, hipStream_t stream){
        axpb_xtended<<<this->num_blocks, this->block_size, 0, stream>>>(
            -alpha,
            this->vec_in[p_idx],
            this->spmv_vec_out[p_idx],
            0,
            this->vec_in[p_idx],
            this->vec_next[p_idx],
            this->coo_partitions[p_idx]->N,
            this->offsets[p_idx],
            0
        );
    });

    for(u32 i = 0; i < this->num_eigencomponents; ++i){

        this->launch_multi_kernel([&](u32 p_idx, hipStream_t stream){
            dot_product_stage_one<<<DOT_PRODUCT_NUM_BLOCKS, this->block_size * (this->num_blocks / DOT_PRODUCT_NUM_BLOCKS), 4 * this->block_size * (this->num_blocks / DOT_PRODUCT_NUM_BLOCKS), stream>>>(
                    this->vec_next[p_idx],
                    this->vec_next[p_idx],
                    this->intermediate_dot_product_values[p_idx],
                    this->coo_partitions[p_idx]->N,
                    0
            );
            dot_product_stage_two<<<1, 32, 0, stream>>>(
                    this->intermediate_dot_product_values[p_idx],
                    &this->beta_intermediate[p_idx]
            );
        });

        hipDeviceSynchronize();

        hipMemcpy(beta_storage_host, this->beta_intermediate, this->num_gpus, hipMemcpyDeviceToHost);
        beta = std::accumulate(beta_storage_host, beta_storage_host + this->num_gpus, 0.0f);
        tridiagonal_matrix.push_back(beta);

        this->launch_multi_kernel([&, beta](u32 p_idx, hipStream_t stream){
                normalize<<<this->num_blocks, this->block_size, 0, stream>>>(
                        this->vec_next[p_idx],
                        1.0f / beta,
                        this->normalized_out[p_idx],
                        this->coo_partitions[p_idx]->N
                );
            });

        this->launch_multi_kernel([&, i](u32 p_idx, hipStream_t stream){
            copy_partition_to_vec<<<this->num_blocks, this->block_size>>>(
                        this->vec_in[p_idx],
                        this->lanczos_vectors[p_idx],
                        this->coo_partitions[p_idx]->N,
                        this->coo_partitions[p_idx]->N * (i - 1),
                        this->offsets[p_idx]
            );
        });

        for(u32 j = 0; j < this->num_gpus; ++j){

            this->launch_multi_kernel([&, i](u32 p_idx, hipStream_t stream){
                copy_partition_to_vec<<<this->num_blocks, this->block_size>>>(
                        this->normalized_out[p_idx],
                        this->vec_in[p_idx],
                        this->coo_partitions[p_idx]->N,
                        offsets[p_idx],
                        0
                );
            });

            auto first = this->vec_in.front();
            this->vec_in.erase(this->vec_in.begin());
            this->vec_in.push_back(first);
        }

        this->launch_multi_kernel([&](u32 p_idx, hipStream_t stream){
            spmv<<<this->num_blocks, this->block_size, 0, stream>>>(
                    this->coo_partitions[p_idx]->x,
                    this->coo_partitions[p_idx]->y,
                    this->coo_partitions[p_idx]->val,
                    this->vec_in[p_idx],
                    this->spmv_vec_out[p_idx],
                    this->coo_partitions[p_idx]->nnz
            );
        });

        this->launch_multi_kernel([&](u32 p_idx, hipStream_t stream){
            dot_product_stage_one<<<DOT_PRODUCT_NUM_BLOCKS, this->block_size * (this->num_blocks / DOT_PRODUCT_NUM_BLOCKS), 0, stream>>>(
                    this->vec_in[p_idx],
                    this->spmv_vec_out[p_idx],
                    this->intermediate_dot_product_values[p_idx],
                    this->coo_partitions[p_idx]->N,
                    this->offsets[p_idx]
            );
            dot_product_stage_two<<<1, 32, 0, stream>>>(
                    this->intermediate_dot_product_values[p_idx],
                    &this->alpha_intermediate[p_idx]
            );
        });

        hipMemcpy(alpha_storage_host, this->alpha_intermediate, this->num_gpus, hipMemcpyDeviceToHost);
        alpha = std::accumulate(alpha_storage_host, alpha_storage_host + this->num_gpus, 0.0f);
        tridiagonal_matrix.push_back(alpha);

        hipDeviceSynchronize();

        this->launch_multi_kernel([&, alpha, beta, i](u32 p_idx, hipStream_t stream){

            axpb_xtended<<<this->num_blocks, this->block_size, 0, stream>>>(
                    -alpha,
                    this->vec_in[p_idx],
                    this->spmv_vec_out[p_idx],
                    -beta,
                    this->lanczos_vectors[p_idx],
                    this->vec_next[p_idx],
                    this->coo_partitions[p_idx]->N,
                    this->offsets[p_idx],
                    this->coo_partitions[p_idx]->N * (i - 1)
            );
        });


        if(this->reorthogonalize){

            for(u32 j = 0; j < i; ++j){
                this->launch_multi_kernel([&, j](u32 p_idx, hipStream_t stream){
                    dot_product_stage_one<<<DOT_PRODUCT_NUM_BLOCKS, this->block_size * (this->num_blocks / DOT_PRODUCT_NUM_BLOCKS), 0, stream>>>(
                            this->vec_next[p_idx],
                            this->lanczos_vectors[p_idx],
                            this->intermediate_dot_product_values[p_idx],
                            this->coo_partitions[p_idx]->N,
                            this->offsets[p_idx] * j
                    );
                    dot_product_stage_two<<<1, 32, 0, stream>>>(
                            this->intermediate_dot_product_values[p_idx],
                            &this->alpha_intermediate[p_idx]
                    );
                });

                alpha = std::accumulate(this->alpha_intermediate, this->alpha_intermediate + this->num_gpus, 0.0f);

                this->launch_multi_kernel([&, alpha](u32 p_idx, hipStream_t stream){
                    subtract<<<this->num_blocks, this->block_size, 0, stream>>>(
                            this->vec_next[p_idx],
                            this->lanczos_vectors[p_idx],
                            alpha,
                            this->coo_partitions[p_idx]->N,
                            this->coo_partitions[p_idx]->N
                    );
                });

            }

        }

        hipDeviceSynchronize();

    }

}

void Benchmark12::execute_sync(i32 iter) {
    assert(this->policy == Policy::Sync);
    this->execute(iter);
}

void Benchmark12::execute_async(int iter) {
    assert(this->policy == Policy::Async);

    for(u32 i = 0; i < this->num_gpus; ++i)
        assert(this->streams[i] != nullptr);

    this->execute(iter);
}

std::string Benchmark12::print_result(bool short_form = false) {
   return "";
}

void Benchmark12::init() {
    // Initialize vec_in[0]
    std::generate(this->vec_in[0], this->vec_in[0] + this->matrix.N, std::rand);
    f32 norm = std::sqrt(std::accumulate(this->vec_in[0], this->vec_in[0] + this->matrix.N, 0.0f, [](f32 acc, f32 cur){
        return acc + cur * cur;
    }));


    // Normalize it
    for(u32 i = 0; i < this->matrix.N; ++i){
        this->vec_in[0][i] /= norm;
    }


    // copy it to the other vectors
    for(u32 i = 1; i < this->num_gpus; ++i){
        hipMemcpy(this->vec_in[i], this->vec_in[0], this->matrix.N, hipMemcpyHostToHost);
    }

    // Initialize the other vectors that get
    // both read and written in a single computation
    for(u32 i = 0; i < this->num_gpus; ++i){
        const auto& partition = this->coo_partitions[i];

        for(u32 j = 0; j < partition->N; ++j){
            this->spmv_vec_out[i][j]    = 0.0f;
            this->vec_next[i][j]        = 0.0f;
            this->normalized_out[i][j]  = 0.0f;
        }

    }

}

void Benchmark12::execute_cudagraph(int iter) {
    throw new std::runtime_error("Benchmark12::execute_cudagraph not implemented");
}

void Benchmark12::execute_cudagraph_manual(int iter) {
    throw new std::runtime_error("Benchmark12::execute_cudagraph_manual not implemented");
}

void Benchmark12::execute_cudagraph_single(int iter) {
    throw new std::runtime_error("Benchmark12::execute_cudagraph_single not implemented");
}

std::ostream &operator<<(std::ostream &os, const coo_matrix_t &matrix) {
    os << "x: " << matrix.x << " y: " << matrix.y << " val: " << matrix.val << " begin: " << matrix.begin << " end: "
       << matrix.end << " N: " << matrix.N << " nnz: " << matrix.nnz;
    return os;
}
