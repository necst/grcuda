#include "hip/hip_runtime.h"
// Copyright (c) 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "b9.cuh"

//////////////////////////////
//////////////////////////////

#define P 1
#define ITER 10

// z = x @ y;
extern "C" __global__ void matrix_vector_mult(const float* x, const float* y, float* z, int n, int m, int z_offset) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        float sum = 0;
        for (int j = 0; j < m; j++) {                
            sum += x[i * m + j] * y[j];
        }
        z[z_offset + i] = sum;
    }
}

// z := w + alpha * A @ y;
extern "C" __global__ void matrix_vector_mult_axpy(const float* x, const float* y, const float *w, const float alpha, float* z, int n, int m, int z_offset) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        float sum = 0;
        for (int j = 0; j < m; j++) {                
            sum += x[i * m + j] * y[j];
        }
        z[z_offset + i] = alpha * sum + w[z_offset + i];
    }
}

__inline__ __device__ float warp_reduce(float val) {
    int warp_size = 32;
    for (int offset = warp_size / 2; offset > 0; offset /= 2) 
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

// z = <x, x>;
extern "C" __global__ void l2_norm(const float *x, float* z, int N) {
    int warp_size = 32;
    float sum = float(0);
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        float x_tmp = x[i];
        sum += x_tmp * x_tmp;
    }
    sum = warp_reduce(sum); // Obtain the sum of values in the current warp;
    if ((threadIdx.x & (warp_size - 1)) == 0) // Same as (threadIdx.x % warp_size) == 0 but faster
        atomicAdd(z, sum); // The first thread in the warp updates the output;
}

// z = <x, y>;
extern "C" __global__ void dot(const float *x, const float *y, float* z, int N) {
    int warp_size = 32;
    float sum = float(0);
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        sum += x[i] * y[i];
    }
    sum = warp_reduce(sum); // Obtain the sum of values in the current warp;
    if ((threadIdx.x & (warp_size - 1)) == 0) // Same as (threadIdx.x % warp_size) == 0 but faster
        atomicAdd(z, sum); // The first thread in the warp updates the output;
}

// y = val + alpha * x;
extern "C" __global__ void saxpy(float* y, float *val, float *x, float alpha, int n) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        y[i] = val[i] + alpha * x[i];
    }
}

// Simply copy array x into y;
extern "C" __global__ void cpy(float *y, const float *x, int n) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        y[i] = x[i];
    }
}

//////////////////////////////
//////////////////////////////

void Benchmark9M::alloc() {
    S = (N + P - 1) / P;
    A = (float **) malloc(sizeof(float*) * P);
    for (int i = 0; i < P; i++) {
        err = hipMallocManaged(&A[i], sizeof(float) * S * N);
    }
    err = hipMallocManaged(&x, sizeof(float) * N);
    err = hipMallocManaged(&b, sizeof(float) * N);
    err = hipMallocManaged(&p, sizeof(float) * N);
    err = hipMallocManaged(&r, sizeof(float) * N);
    err = hipMallocManaged(&y, sizeof(float) * N);
    err = hipMallocManaged(&t1, sizeof(float));
    err = hipMallocManaged(&t2, sizeof(float));

    // Create streams;
    hipStream_t s1, s2;
    err = hipStreamCreate(&s1);
    err = hipStreamCreate(&s2);
    // Create P streams;
    s = (hipStream_t *) malloc(sizeof(hipStream_t) * P);
    for (int i = 0; i < P; i++) {
        hipSetDevice(select_gpu(i, max_devices));
        err = hipStreamCreate(&s[i]);
    }
}

void Benchmark9M::init() {
    // Random input matrix;
    float max = float(RAND_MAX);
    for (int i = 0; i < P; i++) {
        for (int j = 0; j < S * N; j++) {
            A[i][j] = float(rand()) / max;
        }
    }
    // for (int i = 0; i < N * N; i++) {
    //     A[i] = float(rand()) / max;
    // }
    // Random input b;
    for (int i = 0; i < N; i++) {
        b[i] = float(rand()) / max;
    }
}

void Benchmark9M::reset() {
    // Default init of solution x;
    for (int i = 0; i < N; i++) {
        x[i] = 1.0;
    }
    // Reset norms;
    *t1 = 0.0;
    *t2 = 0.0;

    // for (int i = 0; i < P; i++) {
    //     for (int j = 0; j < S * M; j++) {
    //         x[i][j] = x_cpu[i * S * M + j];
    //     }
    // }
}

void Benchmark9M::execute_sync(int iter) { 

    if (pascalGpu && do_prefetch) {
        for (int i = 0; i < P; i++) {
            hipMemPrefetchAsync(A[i], sizeof(float) * S * N, 0);
        }
        hipMemPrefetchAsync(x, sizeof(float) * N, 0);
        hipMemPrefetchAsync(b, sizeof(float) * N, 0);
        hipMemPrefetchAsync(r, sizeof(float) * N, 0);
        hipMemPrefetchAsync(p, sizeof(float) * N, 0);
    }

    for (int i = 0; i < P; i++) {
        matrix_vector_mult_axpy<<<num_blocks, block_size_1d>>>(A[i], x, b, -1, r, S, N, i * S);
        hipDeviceSynchronize();
    }
    cpy<<<num_blocks, block_size_1d>>>(p, r, N);
    hipDeviceSynchronize();
    l2_norm<<<num_blocks, block_size_1d>>>(r, t1, N);
    hipDeviceSynchronize();
    for (int i = 0; i < ITER; i++) {
        for (int i = 0; i < P; i++) {
            matrix_vector_mult<<<num_blocks, block_size_1d>>>(A[i], p, y, S, N, i * S);
            hipDeviceSynchronize();
        }
        dot<<<num_blocks, block_size_1d>>>(p, y, t2, N);
        hipDeviceSynchronize();
        float alpha = *t1 / *t2;
        float old_t1 = *t1;
        *t1 = 0.0;
        saxpy<<<num_blocks, block_size_1d>>>(x, x, p, alpha, N);
        hipDeviceSynchronize();
        saxpy<<<num_blocks, block_size_1d>>>(r, r, y, -1.0 * alpha, N);
        hipDeviceSynchronize();
        l2_norm<<<num_blocks, block_size_1d>>>(r, t1, N);
        hipDeviceSynchronize();
        float beta = *t1 / old_t1;
        saxpy<<<num_blocks, block_size_1d>>>(p, r, p, beta, N);
        hipDeviceSynchronize();
    }
    hipDeviceSynchronize();
}

void Benchmark9M::execute_async(int iter) {
    if (pascalGpu && do_prefetch) {
        for (int i = 0; i < P; i++) {
            hipSetDevice(select_gpu(i, max_devices));
            hipMemPrefetchAsync(A[i], sizeof(float) * S * N, 0, s[i]);
        }
        hipSetDevice(select_gpu(0, max_devices));
        hipMemPrefetchAsync(x, sizeof(float) * N, 0, s1);
        hipMemPrefetchAsync(b, sizeof(float) * N, 0, s1);
        hipMemPrefetchAsync(r, sizeof(float) * N, 0, s1);
        hipMemPrefetchAsync(p, sizeof(float) * N, 0, s1);
    }

    hipEvent_t e[P];
    for (int i = 0; i < P; i++) {
        hipSetDevice(select_gpu(i, max_devices));
        matrix_vector_mult_axpy<<<num_blocks, block_size_1d, 0, s[i]>>>(A[i], x, b, -1, r, S, N, i * S);
        hipEventCreate(&e[i]);
        hipEventRecord(e[i], s[i]);
    }
    hipSetDevice(select_gpu(0, max_devices));
    for (int i = 0; i < P; i++) {
        hipStreamWaitEvent(s1, e[i], 0);
    }
    cpy<<<num_blocks, block_size_1d, 0, s1>>>(p, r, N);
    for (int i = 0; i < P; i++) {
        hipStreamWaitEvent(s2, e[i], 0);
    }
    l2_norm<<<num_blocks, block_size_1d, 0, s2>>>(r, t1, N);
    for (int i = 0; i < ITER; i++) {
        hipEvent_t e2[P];
        for (int i = 0; i < P; i++) {
            hipSetDevice(select_gpu(i, max_devices));
            matrix_vector_mult<<<num_blocks, block_size_1d, 0, s[i]>>>(A[i], p, y, S, N, i * S);
            hipEventCreate(&e2[i]);
            hipEventRecord(e2[i], s[i]);
        }
        hipSetDevice(select_gpu(0, max_devices));
        for (int i = 0; i < P; i++) {
            hipStreamWaitEvent(s1, e2[i], 0);
        }
        dot<<<num_blocks, block_size_1d, 0, s1>>>(p, y, t2, N);
        hipStreamSynchronize(s1);
        hipStreamSynchronize(s2);
        float alpha = *t1 / *t2;
        float old_t1 = *t1;
        *t1 = 0.0;
        saxpy<<<num_blocks, block_size_1d, 0, s1>>>(x, x, p, alpha, N);
        saxpy<<<num_blocks, block_size_1d, 0, s2>>>(r, r, y, -1.0 * alpha, N);
        l2_norm<<<num_blocks, block_size_1d, 0, s2>>>(r, t1, N);
        hipStreamSynchronize(s2);
        float beta = *t1 / old_t1;
        saxpy<<<num_blocks, block_size_1d, 0, s1>>>(p, r, p, beta, N);
    }
    hipStreamSynchronize(s1);
}

std::string Benchmark9M::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(x[0]);
    } else {
        std::string res = "[";
        for (int j = 0; j < std::min(10, N); j++) {
            res += std::to_string(x[j]) + ", ";
        }

        float sum = 0;
        for (int j = 0; j < N; j++) {
            sum += x[j];
        }
        return res + "...], sum=" + std::to_string(sum);
    }
}