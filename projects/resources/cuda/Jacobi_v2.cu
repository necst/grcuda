
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define N 10000
#define IT 500

__global__ void JacobiIteration(int n, float *a, float *b, float *x, float*x_result){
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){
        float sigma = 0;
        for(int j = 0 ; j<n; j++){
            if(j!=i){
                sigma += a[j + i * n]*x[j];
            }
        }
        x_result[i] = (b[i] - sigma)/a[i + i*n];
    }
}
__global__ void JacobiIterationDistributed(int n, float *a, float *x, float*sigma){
    for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < n*n; tid += blockDim.x * gridDim.x){
        int i = tid/n;
        int j = tid%n;
        if(j!=i){
            atomicAdd(&sigma[i],a[tid]*x[j]);
        }
    }
}
__global__ void JacobiIterationDistributedResult(int n, float *a, float *b, float*x_result, float*sigma){
    for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < n; tid += blockDim.x * gridDim.x){
        x_result[tid] = (b[tid] - sigma[tid])/a[tid + tid*n];
        sigma[tid] = 0;
    }
}
__global__ void setSigmaToZero(int n, float*sigma){
    for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < n; tid += blockDim.x * gridDim.x){
       sigma[tid] = 0;
    }
}

__global__ void initAMatrix(int n, float*a){
    int i;
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n; j += blockDim.x * gridDim.x){
    
        for ( i = 0; i < n; i++ ){
            if ( j == i - 1 ){
                a[j+i*n] = -1.0;
            }
            else if ( j == i ){
                a[j+i*n] = 2.0;
            }
            else if ( j == i + 1 ){
                a[j+i*n] = -1.0;
            }
            else{
                a[j+i*n] = 0.0;
            }
        }
    }
      
}

__global__ void copy(int n, float*a, float *b){
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n; j += blockDim.x * gridDim.x){
        a[j] = b[j];
    }
      
}

void swap(float* &a, float* &b){
    float *temp = a;
    a = b;
    b = temp;
  }
  


int main(){
    float *a, *b, *x, *x_result,*sigma;

    // alloc
    hipMallocManaged(&a, N*N*sizeof(float));
    hipMallocManaged(&b, N*sizeof(float));
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&x_result, N*sizeof(float));
    hipMallocManaged(&sigma, N*sizeof(float));
    // init

    for (int i = 0; i < N; i++ )
    {
        sigma[i] = 0;
    }

    for (int i = 0; i < N; i++ )
    {
        b[i] = 3.0;
    }
    b[N-1] = ( float ) ( N + 1 );

    for ( int i = 0; i < N; i++ )
    {
      x[i] = 0.0;
    }

    initAMatrix<<<32, 32>>>(N, a);

    float elapsed=0;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    for ( int it = 0; it < IT; it++ ){        
        JacobiIteration<<<32, 32>>>(N, a, b, x, x_result);
        hipDeviceSynchronize();
        swap(x, x_result);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize (stop);

    hipEventElapsedTime(&elapsed, start, stop); 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("execution time: %f \n", elapsed);

    printf("-------------------------------------------------------\n");
    // for(int i = 0; i < N; i++){
    //     printf("%f ",x[i]);
    // }

    // printf("\n");

    for (int i = 0; i < N; i++ )
    {
        sigma[i] = 0;
    }

    for (int i = 0; i < N; i++ )
    {
        b[i] = 3.0;
    }
    b[N-1] = ( float ) ( N + 1 );

    for ( int i = 0; i < N; i++ )
    {
      x[i] = 0.0;
    }

    initAMatrix<<<32, 32>>>(N, a);


    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    for ( int it = 0; it < IT; it++ ){  
        JacobiIterationDistributed<<<896, 32>>>(N, a, x, sigma);
        hipDeviceSynchronize();
        JacobiIterationDistributedResult<<<896, 32>>>(N, a, b, x_result, sigma);
        hipDeviceSynchronize();
        swap(x, x_result);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize (stop );

    hipEventElapsedTime(&elapsed, start, stop); 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("execution time: %f \n", elapsed);
    
    printf("-------------------------------------------------------\n");
    // for(int i = 0; i < N; i++){
    //     printf("%f ",x[i]);
    // }
    // printf("\n");

    return 0;
}