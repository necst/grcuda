
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define N 100
#define IT 3

__global__ void JacobiIteration(int n, float *a, float *b, float *x, float*x_result){
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){
        float sigma = 0;
        for(int j = 0 ; j<n; j++){
            if(j!=i){
                sigma += a[i + j * n]*x[j];
            }
        }
        x_result[i] = (b[i] - sigma)/a[i + i*n];
    }
}

__global__ void initAMatrix(int n, float*a){
    int i;
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n; j += blockDim.x * gridDim.x){
    
        for ( i = 0; i < n; i++ ){
            if ( j == i - 1 ){
                a[j+i*n] = -1.0;
            }
            else if ( j == i ){
                a[j+i*n] = 2.0;
            }
            else if ( j == i + 1 ){
                a[j+i*n] = -1.0;
            }
            else{
                a[j+i*n] = 0.0;
            }
        }
    }
      
}

__global__ void copy(int n, float*a, float *b){
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n; j += blockDim.x * gridDim.x){
        a[j] = b[j];
    }
      
}

void swap(float* &a, float* &b){
    float *temp = a;
    a = b;
    b = temp;
  }
  


int main(){
    float *a, *b, *x, *x_result;

    // alloc
    hipMallocManaged(&a, N*N*sizeof(float));
    hipMallocManaged(&b, N*sizeof(float));
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&x_result, N*sizeof(float));

    // init
    for (int i = 0; i < N; i++ )
    {
        b[i] = 3.0;
    }
    b[N-1] = ( float ) ( N + 1 );

    for ( int i = 0; i < N; i++ )
    {
      x[i] = 0.0;
    }

    initAMatrix<<<32, 32>>>(N, a);

    for ( int it = 0; it < IT; it++ ){        

        JacobiIteration<<<32, 32>>>(N, a, b, x, x_result);
        hipDeviceSynchronize();
        swap(x, x_result);

    }

    for(int i = 0; i < N; i++){
        printf("%f ",x[i]);
    }
    return 0;
}