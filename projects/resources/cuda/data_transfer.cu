
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

float p2p_copy (size_t size)
{
  int *pointers[2];

  hipSetDevice (0);
  hipDeviceEnablePeerAccess (1, 0);
  hipMalloc (&pointers[0], size);

  hipSetDevice (1);
  hipDeviceEnablePeerAccess (0, 0);
  hipMalloc (&pointers[1], size);

  hipEvent_t begin, end;
  hipEventCreate (&begin);
  hipEventCreate (&end);

  hipEventRecord (begin);
  hipMemcpyAsync (pointers[0], pointers[1], size, hipMemcpyDeviceToDevice);
  hipEventRecord (end);
  hipEventSynchronize (end);

  float elapsed;
  hipEventElapsedTime (&elapsed, begin, end);
  elapsed /= 1000;

  hipSetDevice (0);
  hipFree (pointers[0]);

  hipSetDevice (1);
  hipFree (pointers[1]);

  hipEventDestroy (end);
  hipEventDestroy (begin);

  return elapsed;
}

void printDeviceAttribute(){
    int attr_val_device_0 = 0;
    int attr_val_device_1 = 0;
    hipError_t err;
    hipDeviceGetAttribute(&attr_val_device_0,hipDeviceAttributeConcurrentManagedAccess, 0);
    hipDeviceGetAttribute(&attr_val_device_1,hipDeviceAttributeConcurrentManagedAccess, 1);

    printf("concurrent managed access device 0: %d \nconcurrent managed access device 1: %d\n", attr_val_device_0, attr_val_device_1);

    int can_access_peer_device_0 = 0;
    int can_access_peer_device_1 = 0;
    hipDeviceCanAccessPeer(&can_access_peer_device_0,0,1);
    hipDeviceCanAccessPeer(&can_access_peer_device_1,1,0);
    printf("concurrent peer access device 0->1: %d \nconcurrent peer access device 1->0: %d\n", can_access_peer_device_0, can_access_peer_device_1);

    hipSetDevice(0);
    err = hipDeviceEnablePeerAccess(1, 0);
    printf("err: %s\n", hipGetErrorString(err));

    hipSetDevice(1);
    err = hipDeviceEnablePeerAccess(0, 0);
    printf("err: %s\n", hipGetErrorString(err));

    // cudaSetDevice(0);
    // err = cudaDeviceDisablePeerAccess(1);
    // printf("err: %s\n", cudaGetErrorString(err));

    // cudaSetDevice(1);
    // err = cudaDeviceDisablePeerAccess(0);
    // printf("err: %s\n", cudaGetErrorString(err));


    // cudaDeviceCanAccessPeer(&can_access_peer_device_0,0,1);
    // cudaDeviceCanAccessPeer(&can_access_peer_device_1,1,0);
    // printf("concurrent peer access device 0->1: %d \nconcurrent peer access device 1->0: %d\n", can_access_peer_device_0, can_access_peer_device_1);


}


int main(){
    size_t size = 1000000000; // 100_000_000_000
    float time_first = p2p_copy(size);
    printf("time spend %f \n",time_first);

    float time_second = p2p_copy(size);
    printf("time spend %f \n",time_second);
    printDeviceAttribute();
    float time_third = p2p_copy(size);
    printf("time spend %f \n",time_third);
    return 0;
}